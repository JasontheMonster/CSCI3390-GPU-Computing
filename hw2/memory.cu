#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "timerc.h"


#define gerror(ans) { gpuAssert ((ans), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{ 
  if (code !=hipSuccess)
  {
    fprintf(stderr, "GPUassert:%s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


__device__ int a[1];

__global__ void printfaddress(int *r){
  printf("Content of address %p from GPU = %d\n", r, r[0]);
}

__global__ void increase_memory(int *r, int n){
  int index = threadIdx.x + blockDim.x*blockIdx.x;

  if(index <n){
    r[index] = r[index] +index;
  }
}


int main(void){
  hipSetDevice(0);


  int *dev_ptr;
  int n = 128;

  int *host_ptr = (int *) malloc(n*sizeof(int));
  for (int i = 0; i <n; i++){
    host_ptr[i] = i;
  }

  hipMalloc((void**)&dev_ptr, sizeof(int)*n);
  hipMemcpy(dev_ptr, host_ptr, sizeof(int)*n, hipMemcpyHostToDevice);

  dim3 numthreadsperblock (1024, 1);
  dim3 numblockspergrid((n+1023)/1024, 1);

  increase_memory<<<numblockspergrid, numthreadsperblock>>>(dev_ptr, n);

  hipMemcpy( host_ptr, dev_ptr, sizeof(int)*n, hipMemcpyDeviceToHost);

  for (int i =0; i <n; i++){
    printf("%d ", host_ptr[i]);
  }
  printf("\n");
  free(host_ptr);
  hipFree(dev_ptr); 

  int a_host = 66;

  printf("Address of a from CPUs = %p\n", a);

  int *address_of_a;
  hipGetSymbolAddress(( void **) &address_of_a, a);

  printf("Address of a from GPU = %p\n", address_of_a);

  hipMemcpyToSymbol(HIP_SYMBOL(a), &a_host, sizeof(int), 0, hipMemcpyHostToDevice);

  printf("Content of address from GPU = %d\n", a[0]);

  printfaddress<<<1, 2>>>( address_of_a);
  hipDeviceSynchronize();

  gerror( hipPeekAtLastError() );
  hipDeviceSynchronize();


}
